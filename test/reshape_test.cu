#include "hip/hip_runtime.h"
#include <cutt/reshape.hpp>
#include <iostream>
#include <vector>
#include <algorithm>

template <class T>
void reshape_test(
		const std::vector<unsigned> mode_dim
		) {
	std::vector<std::pair<std::string, std::size_t>> mode;
	std::vector<std::string> original_mode;
	for (unsigned i = 0; i < mode_dim.size(); i++) {
		const auto name = "m" + std::to_string(i);
		mode.push_back(std::make_pair(name, mode_dim[i]));

		original_mode.push_back(name);
	}
	std::vector<std::string> reshaped_mode(mode.size());
	std::reverse_copy(original_mode.begin(), original_mode.end(), reshaped_mode.begin());

	std::size_t dim_product = 1;
	for (const auto d : mode_dim) {
		dim_product *= d;
	}

	T* original_ptr;
	hipMallocManaged(&original_ptr, sizeof(T) * dim_product);
	const auto mom = std::sqrt(static_cast<T>(dim_product));
	for (std::size_t i = 0; i < dim_product; i++) {
		original_ptr[i] = i / mom;
	}

	T* reshaped_ptr;
	hipMallocManaged(&reshaped_ptr, sizeof(T) * dim_product);

	cutt::reshape(
			original_ptr,
			reshaped_ptr,
			mode,
			reshaped_mode
			);

	hipFree(original_ptr);
	hipFree(reshaped_ptr);
}

int main() {
	reshape_test<float>({1000, 1000, 1000});
}
