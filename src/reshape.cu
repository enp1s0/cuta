#include "hip/hip_runtime.h"
#include <cutt/reshape.hpp>
#include "utils.hpp"
#include <hip/hip_fp16.h>
#include <unordered_map>

namespace {
constexpr unsigned max_num_mode = 30;
__constant__ std::size_t c_reshaped_stride[max_num_mode];
__constant__ std::size_t c_reshaped_dim   [max_num_mode];

template <class T>
__global__ void reshpae_kernel (
		T* const dst_ptr,
		const T* const src_ptr,
		const unsigned num_mode,
		const std::size_t num_elements
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= num_elements) {
		return;
	}

	const auto v = src_ptr[tid];

	auto dst_j = tid;
	auto dst_i = decltype(tid)(0);
	for (unsigned i = 0; i < num_mode; i++) {
		dst_i += (dst_j % c_reshaped_dim[i]) * c_reshaped_stride[i];
		dst_j /= c_reshaped_dim[i];
	}
	dst_ptr[dst_i] = v;
}
} // noname namespace

template <class T>
void cutt::reshape(
		T *const dst_ptr,
		const T* const src_ptr,
		const std::vector<std::pair<std::string, std::size_t>>& mode,
		const std::vector<std::string>& reshaped_order,
		hipStream_t hip_stream) {

	const auto num_mode = mode.size();

	// Validations
	if (num_mode > max_num_mode) {
		throw std::runtime_error("The maximum number of modes is " + std::to_string(max_num_mode) + ". Given " + std::to_string(num_mode) + ".");
	}

	if (num_mode != reshaped_order.size()) {
		throw std::runtime_error("The size of reshaped mode order list is different from mode list.");
	}

	// Calculate strides
	std::unordered_map<std::string, std::size_t> stride;
	std::size_t dim_product = 1;
	for (const auto& m : mode) {
		stride.insert(std::make_pair(m.first, dim_product));
		dim_product *= m.second;
	}

	std::vector<std::size_t> reshaped_stride(num_mode);
	std::vector<std::size_t> reshaped_dim   (num_mode);

	for (unsigned i = 0; i < num_mode; i++) {
		reshaped_stride[i] = stride[reshaped_order[i]];
		for (const auto& m : mode) {
			if (m.first == reshaped_order[i]) {
				reshaped_dim   [i] = m.second;
			}
		}
	}

	CUTT_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_reshaped_stride), reshaped_stride.data(), sizeof(std::size_t) * num_mode, 0, hipMemcpyHostToDevice, hip_stream));
	CUTT_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_reshaped_dim   ), reshaped_dim   .data(), sizeof(std::size_t) * num_mode, 0, hipMemcpyHostToDevice, hip_stream));

	const unsigned block_size = 256;
	const auto grid_size = (dim_product + block_size - 1) / block_size;

	reshpae_kernel<<<grid_size, block_size, 0, hip_stream>>>(
			dst_ptr,
			src_ptr,
			num_mode,
			dim_product
			);
}

#define CUTT_RESHAPE_INSTANCE(type) \
template void cutt::reshape<type>(type* const, const type* const, const std::vector<std::pair<std::string, std::size_t>>&, const std::vector<std::string>&, hipStream_t);
CUTT_RESHAPE_INSTANCE(double);
CUTT_RESHAPE_INSTANCE(float );
CUTT_RESHAPE_INSTANCE(half  );
